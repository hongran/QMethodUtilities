#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>

#include "TFile.h"
#include "TH1.h"

#include "RPAnalysisModule.h"

// Analysis device kernel
__global__ void flush_analysis(float *FlushQArray, float *AnaQArray,
                               float *AnaPedArray, float *S_ADCHistArray,
                               float *C_ADCHistArray,
                               int *AnalysisIntParameters,
                               float *AnalysisParameters) {
  int flush_buffer_max_length = AnalysisIntParameters[1];
  int nFlushesPerBatch = AnalysisIntParameters[0];
  int wndw = AnalysisIntParameters[2];
  int gap = AnalysisIntParameters[3];
  int EBinW = AnalysisIntParameters[4];
  // Low energy cut in energy in ADC histogram.
  int lowECut = AnalysisIntParameters[5];

  float threshold = AnalysisParameters[0];

  // thread index
  int iflush = blockIdx.x * blockDim.x + threadIdx.x;

  if (iflush < nFlushesPerBatch) {
    int flushoffset = iflush * NSEG * flush_buffer_max_length;
    for (int iADC = gap + wndw; iADC < flush_buffer_max_length - gap - wndw - 1;
         iADC++) {
      float sig_sum = 0;
      for (int iseg = 0; iseg < NSEG; iseg++) {
        int segmentoffset = iseg * flush_buffer_max_length;
        int idx = flushoffset + segmentoffset + iADC;
        float InputBuffer[32];
        int BufferStartIdx = iADC - gap - wndw;
        for (int kADC = iADC - gap - wndw; kADC < iADC + gap + wndw + 1;
             kADC++) {
          InputBuffer[kADC - BufferStartIdx] = FlushQArray[kADC - iADC + idx];
        }

        float ysum = 0, yavg = 0;
        // find the mask base on rejection logic: yi - sum(y k!=i)/5 >
        // threshold
        int mask[8] = {1, 1, 1, 1, 1, 1, 1, 1};
        // For samples in the window region left to the trigger sample
        for (int jADC = 0; jADC < wndw; jADC++) {
          ysum = 0;
          for (int kADC = iADC - gap - wndw; kADC < iADC + gap + wndw + 1;
               kADC++) {
            if (kADC != jADC + iADC - wndw - gap) {
              if (kADC - iADC + gap < 0)
                // ysum += FlushQArray[kADC - iADC + idx];
                ysum += InputBuffer[kADC - BufferStartIdx];

              if (kADC - iADC - gap > 0)
                // ysum += FlushQArray[kADC - iADC + idx];
                ysum += InputBuffer[kADC - BufferStartIdx];
            }
          }

          yavg = ysum / (2.0 * wndw - 1);

          if (InputBuffer[jADC] - yavg > threshold) {
            mask[jADC] = 0;
          }
        } // End of samples in the window region left to the trigger sample.

        // For samples in the window region right to the trigger sample.
        for (int jADC = wndw; jADC < 2 * wndw; jADC++) {
          ysum = 0;
          for (int kADC = iADC - gap - wndw; kADC < iADC + gap + wndw + 1;
               kADC++) {
            if (kADC != jADC + iADC - wndw + gap + 1) {
              if (kADC - iADC + gap < 0)
                // ysum += FlushQArray[kADC - iADC + idx];
                ysum += InputBuffer[kADC - BufferStartIdx];

              if (kADC - iADC - gap > 0)
                // ysum += FlushQArray[kADC - iADC + idx];
                ysum += InputBuffer[kADC - BufferStartIdx];
            }
          }

          yavg = ysum / (2.0 * wndw - 1);

          if (InputBuffer[jADC + 2 * gap + 1] - yavg > threshold) {
            mask[jADC] = 0;
          }
        } // End of samples in the window region right to the trigger sample.

        // compute the pileup corrected pedestal
        ysum = 0;

        for (int jADC = 0; jADC < wndw; jADC++) {
          // ysum += FlushQArray[idx + jADC - gap - wndw] * mask[jADC];
          ysum += InputBuffer[jADC] * mask[jADC];
        }
        for (int jADC = wndw; jADC < 2 * wndw; jADC++) {
          // ysum += FlushQArray[idx + jADC + gap - wndw + 1] * mask[jADC];
          ysum += InputBuffer[jADC + 2 * gap + 1] * mask[jADC];
        }

        yavg = ysum / (2.0 * wndw - 1);
        // float ydiff = FlushQArray[idx] - yavg;
        float ydiff = InputBuffer[wndw + gap] - yavg;
        AnaPedArray[idx] = yavg;

        if (ydiff > threshold) {
          AnaQArray[idx] += ydiff;
          uint s_binIdx = __float2uint_rd((ydiff - lowECut) / EBinW);
          atomicAdd(&S_ADCHistArray[s_binIdx], 1);
          sig_sum += ydiff;
        }
      }
      uint c_binIdx = __float2uint_rd((sig_sum - lowECut) / EBinW);
      atomicAdd(&C_ADCHistArray[c_binIdx], 1);
    }
  }
}

namespace QAnalysis {
RPAnalysisModule::RPAnalysisModule(
    std::string Name, const std::map<std::string, int> &tIntParameters,
    const std::map<std::string, float> &tFloatParameters,
    const std::map<std::string, std::string> &tStringParameters,
    int nFlushesPerBatch, int FillMaxLength)
    : AnalysisModule(Name, tIntParameters, tFloatParameters,
                     tStringParameters) {
  // Initialize the parameter arrays
  IntParameters["NFlushesPerBatch"] = nFlushesPerBatch;
  IntParameters["FillBufferMaxLength"] = FillMaxLength;
  InitParameters();
  // hipSetDevice(1);

  // Allocate Derive memory for parameters

  hipMalloc((void **)&d_AnalysisParameters,
             AnalysisParameters.size() * sizeof(float));
  hipMalloc((void **)&d_AnalysisIntParameters,
             AnalysisIntParameters.size() * sizeof(int));

  hipMemcpy(d_AnalysisParameters, &AnalysisParameters[0],
             AnalysisParameters.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_AnalysisIntParameters, &AnalysisIntParameters[0],
             AnalysisIntParameters.size() * sizeof(float),
             hipMemcpyHostToDevice);

  // Arrays
  ArraySizes["AnaQArray"] =
      nFlushesPerBatch * NSEG * FillMaxLength * sizeof(float);
  ArraySizes["AnaPedArray"] =
      nFlushesPerBatch * NSEG * FillMaxLength * sizeof(float);
  ArraySizes["RPSegmADCHistArray"] = 1000 * sizeof(float);
  ArraySizes["RPCaloADCHistArray"] = 1000 * sizeof(float);
  // Allocate memories
  for (auto it = ArraySizes.begin(); it != ArraySizes.end(); ++it) {
    auto Name = it->first;
    auto Size = it->second;
    HostArrays[Name] = (float *)malloc(Size);
    hipMalloc((void **)&DeviceArrays[Name], Size);
  }
}

// Analysis Functions
int RPAnalysisModule::FlushAnalysis(
    std::map<std::string, float *> *SimulatorHostArrays,
    std::map<std::string, float *> *SimulatorDeviceArrays,
    std::map<std::string, int> *SimulatorArraySizes) {

  int nblocks =
      IntParameters["NFlushesPerBatch"] / IntParameters["NThreadsPerBlock"] + 1;
  std::cout << "Analyzing flush batch" << std::endl;

  float time;
  hipEvent_t start, stop;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  flush_analysis<<<nblocks, IntParameters["NThreadsPerBlock"]>>>(
      (*SimulatorDeviceArrays)["FlushQArray"], DeviceArrays["AnaQArray"],
      DeviceArrays["AnaPedArray"], DeviceArrays["RPSegmADCHistArray"], 
      DeviceArrays["RPCaloADCHistArray"],d_AnalysisIntParameters,
      d_AnalysisParameters);

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

  printf("Time to Analyze flush:  %3.1f ms \n", time);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Cuda failure with user kernel function make flush anlaysis %s:%d: "
           "'%s'\n",
           __FILE__, __LINE__, hipGetErrorString(err));
    exit(0);
  }

  return 0;
}

int RPAnalysisModule::EndAnalysis(
    std::map<std::string, float *> *SimulatorHostArrays,
    std::map<std::string, int> *SimulatorArraySizes) {
  // copy back to host
  int n = 0;
  hipError_t err;
  for (auto it = ArraySizes.begin(); it != ArraySizes.end(); ++it) {
    auto Name = it->first;
    auto Size = it->second;
    hipMemcpy(HostArrays[Name], DeviceArrays[Name], Size,
               hipMemcpyDeviceToHost);
    //      std::cout<< n << " "<<Name<<" "<<Size<<std::endl;
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Cuda failure with user kernel function make)analysis endrun copy "
             "%s:%d: '%s'\n",
             __FILE__, __LINE__, hipGetErrorString(err));
      exit(0);
    }
    n++;
  }

  return 0;
}

int RPAnalysisModule::Output(int RunNumber) {
  std::vector<double> AnaQHist;
  std::vector<double> AnaPedHist;
  this->GetCaloArray("AnaQArray", AnaQHist);
  this->GetCaloArray("AnaPedArray", AnaPedHist, false);

  unsigned int N = AnaQHist.size();

  TH1 *hAnaQ = new TH1D("AnaQHist", "AnaQHist", N, 0, N * 0.075);
  for (unsigned int i = 0; i < N; i++) {
    hAnaQ->SetBinContent(i, AnaQHist[i]);
  }

  TH1 *hAnaPed = new TH1D("AnaPedHist", "AnaPedHist", N, 0, N * 0.075);
  for (unsigned int i = 0; i < N; i++) {
    hAnaPed->SetBinContent(i, AnaPedHist[i]);
  }

  auto ptr1 = HostArrays["RPSegmADCHistArray"];
  auto ptr2 = HostArrays["RPCaloADCHistArray"];
  auto Size = ArraySizes["RPSegmADCHistArray"] / sizeof(float);
  int ADCBinW = IntParameters["ADCBinWidth"];
  int lowADCCut = IntParameters["LowADCCut"];

  TH1 *hSegmADC = new TH1D("RP_Segm_ADC_Hist", "Segments ADC Distribution, RP Method", Size,
                           lowADCCut, lowADCCut + Size * ADCBinW);
  for (unsigned int i = 0; i < Size; i++) {
    hSegmADC->SetBinContent(i, ptr1[i]);
  }
  TH1 *hCaloADC = new TH1D("RP_Calo_ADC_Hist", "Calorimeter ADC Distribution, RP Method",
                           Size, lowADCCut, lowADCCut + Size * ADCBinW);
  for (unsigned int i = 0; i < Size; i++) {
    hCaloADC->SetBinContent(i, ptr2[i]);
  }
  TFile *FileOut =
      new TFile(Form("RPRootOut_%04d.root", RunNumber), "recreate");
  hAnaQ->Write();
  hAnaPed->Write();
  hSegmADC->Write();
  hCaloADC->Write();
  FileOut->Close();

  return 0;
}

// Private Functions
int RPAnalysisModule::InitParameters() {
  AnalysisParameters.resize(1);
  AnalysisIntParameters.resize(6);

  AnalysisParameters[0] = FloatParameters["Threshold"];

  AnalysisIntParameters[0] = IntParameters["NFlushesPerBatch"];
  AnalysisIntParameters[1] = IntParameters["FillBufferMaxLength"];
  AnalysisIntParameters[2] = IntParameters["Window"];
  AnalysisIntParameters[3] = IntParameters["Gap"];
  AnalysisIntParameters[4] = IntParameters["ADCBinWidth"];
  AnalysisIntParameters[5] = IntParameters["LowADCCut"];
  return 0;
}

int RPAnalysisModule::GetArray(std::string ArrayName,
                               std::vector<double> &Output) {
  auto Size = ArraySizes[ArrayName];
  Output.resize(Size);
  auto ptr = HostArrays[ArrayName];
  for (int i = 0; i < Size; i++) {
    Output[i] = ptr[i];
  }
  return 0;
}

int RPAnalysisModule::GetCaloArray(std::string ArrayName,
                                   std::vector<double> &Output, bool BatchSum) {
  Output.clear();
  Output.resize(IntParameters["FillBufferMaxLength"], 0.0);
  auto ptr = HostArrays[ArrayName];

  int nFlushesPerBatch = IntParameters["NFlushesPerBatch"];

  for (unsigned int k = 0; k < nFlushesPerBatch; k++) {
    for (unsigned int j = 0; j < NSEG; j++) {
      for (unsigned int i = 0; i < IntParameters["FillBufferMaxLength"]; i++) {
        Output[i] +=
            ptr[(k * NSEG + j) * IntParameters["FillBufferMaxLength"] + i];
      }
    }
    if (!BatchSum) {
      break;
    }
  }

  return 0;
}

} // end namespace QAnalysis
