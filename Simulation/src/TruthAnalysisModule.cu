#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>

#include "TFile.h"
#include "TH1.h"

#include "TruthAnalysisModule.h"

// Analysis device kernel
__global__ void flush_analysis(float *FlushQTruthArray, float *AnaQArray,
                               int *AnalysisIntParameters,
                               float *AnalysisParameters) {
  int flush_buffer_max_length = AnalysisIntParameters[1];
  int nFlushesPerBatch = AnalysisIntParameters[0];
  float threshold = AnalysisParameters[0];

  // thread index
  int iflush = blockIdx.x * blockDim.x + threadIdx.x;

  if (iflush < nFlushesPerBatch) {
    for (int idx = 0; idx < NSEG * flush_buffer_max_length; idx++) {
      int flushoffset = iflush * NSEG * flush_buffer_max_length;
      float qdata= FlushQTruthArray[flushoffset + idx];
      if(qdata > threshold){
        AnaQArray[flushoffset + idx] += qdata;
      }
      /*
      if (iflush > 20000)
      {
	printf("%d\n",idx);
      }
      */
    }
  }
}

namespace QAnalysis {
TruthAnalysisModule::TruthAnalysisModule(
    std::string Name, const std::map<std::string, int> &tIntParameters,
    const std::map<std::string, float> &tFloatParameters,
    const std::map<std::string, std::string> &tStringParameters,
    int nFlushesPerBatch, int FillMaxLength)
    : AnalysisModule(Name, tIntParameters, tFloatParameters,
                     tStringParameters) {
  // Initialize the parameter arrays
  IntParameters["NFlushesPerBatch"] = nFlushesPerBatch;
  IntParameters["FillBufferMaxLength"] = FillMaxLength;
  InitParameters();
  // hipSetDevice(1);

  // Allocate Derive memory for parameters

  hipMalloc((void **)&d_AnalysisParameters,
             AnalysisParameters.size() * sizeof(float));
  hipMalloc((void **)&d_AnalysisIntParameters,
             AnalysisIntParameters.size() * sizeof(int));

  hipMemcpy(d_AnalysisParameters, &AnalysisParameters[0],
             AnalysisParameters.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_AnalysisIntParameters, &AnalysisIntParameters[0],
             AnalysisIntParameters.size() * sizeof(float),
             hipMemcpyHostToDevice);

  // Arrays
  ArraySizes["AnaQArray"] =
      nFlushesPerBatch * NSEG * FillMaxLength * sizeof(float);

  // Allocate memories
  for (auto it = ArraySizes.begin(); it != ArraySizes.end(); ++it) {
    auto Name = it->first;
    auto Size = it->second;
    HostArrays[Name] = (float *)malloc(Size);
    hipMalloc((void **)&DeviceArrays[Name], Size);
  }
}

// Analysis Functions
int TruthAnalysisModule::FlushAnalysis(
    std::map<std::string, float *> *SimulatorHostArrays,
    std::map<std::string, float *> *SimulatorDeviceArrays,
    std::map<std::string, int> *SimulatorArraySizes) {

  int nblocks =
      IntParameters["NFlushesPerBatch"] / IntParameters["NThreadsPerBlock"] + 1;
  std::cout << "Analyzing flush batch" << std::endl;

  flush_analysis<<<nblocks, IntParameters["NThreadsPerBlock"]>>>(
      (*SimulatorDeviceArrays)["FlushQTruthArray"], DeviceArrays["AnaQArray"],
      d_AnalysisIntParameters, d_AnalysisParameters);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Cuda failure with user kernel function make)flush anlaysis %s:%d: "
           "'%s'\n",
           __FILE__, __LINE__, hipGetErrorString(err));
    exit(0);
  }

  return 0;
}

int TruthAnalysisModule::EndAnalysis(
    std::map<std::string, float *> *SimulatorHostArrays,
    std::map<std::string, int> *SimulatorArraySizes) {
  // copy back to host
  int n = 0;
  hipError_t err;
  for (auto it = ArraySizes.begin(); it != ArraySizes.end(); ++it) {
    auto Name = it->first;
    auto Size = it->second;
    hipMemcpy(HostArrays[Name], DeviceArrays[Name], Size,
               hipMemcpyDeviceToHost);
    //      std::cout<< n << " "<<Name<<" "<<Size<<std::endl;
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Cuda failure with user kernel function make)analysis endrun "
             "copy "
             "%s:%d: '%s'\n",
             __FILE__, __LINE__, hipGetErrorString(err));
      exit(0);
    }
    n++;
  }

  return 0;
}

int TruthAnalysisModule::Output(int RunNumber) {
  std::vector<double> AnaQHist;
  this->GetCaloArray("AnaQArray", AnaQHist);

  unsigned int N = AnaQHist.size();

  TH1 *hAnaQ = new TH1D("TruthQHist", "TruthQHist", N, 0, N * 0.075);
  for (unsigned int i = 0; i < N; i++) {
    hAnaQ->SetBinContent(i, AnaQHist[i]);
  }


  TFile *FileOut =
      new TFile(Form("TruthRootOut_%04d.root", RunNumber), "recreate");
  hAnaQ->Write();
  FileOut->Close();

  return 0;
}

// Private Functions
int TruthAnalysisModule::InitParameters() {
  AnalysisParameters.resize(1);
  AnalysisIntParameters.resize(2);

  AnalysisParameters[0] = FloatParameters["Threshold"];
  AnalysisIntParameters[0] = IntParameters["NFlushesPerBatch"];
  AnalysisIntParameters[1] = IntParameters["FillBufferMaxLength"];
  // AnalysisIntParameters[2] = IntParameters["Window"];
  // AnalysisIntParameters[3] = IntParameters["Gap"];

  return 0;
}

int TruthAnalysisModule::GetArray(std::string ArrayName,
                               std::vector<double> &Output) {
  auto Size = ArraySizes[ArrayName];
  Output.resize(Size);
  auto ptr = HostArrays[ArrayName];
  for (int i = 0; i < Size; i++) {
    Output[i] = ptr[i];
  }
  return 0;
}

int TruthAnalysisModule::GetCaloArray(std::string ArrayName,
                                   std::vector<double> &Output, bool BatchSum) {
  Output.clear();
  Output.resize(IntParameters["FillBufferMaxLength"], 0.0);
  auto ptr = HostArrays[ArrayName];

  int nFlushesPerBatch = IntParameters["NFlushesPerBatch"];

  for (unsigned int k = 0; k < nFlushesPerBatch; k++) {
    for (unsigned int j = 0; j < NSEG; j++) {
      for (unsigned int i = 0; i < IntParameters["FillBufferMaxLength"]; i++) {
        Output[i] +=
            ptr[(k * NSEG + j) * IntParameters["FillBufferMaxLength"] + i];
      }
    }
    if (!BatchSum) {
      break;
    }
  }

  return 0;
}

} // end namespace QAnalysis
